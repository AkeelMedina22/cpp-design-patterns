#include "hip/hip_runtime.h"
// #ifdef WITH_CUDA

#include <Eigen/Dense>
#include <iostream>
#include <hip/hip_runtime.h>
#include "logger.hpp"
#include "cuda_multiplier.hpp"
#include "multiplier_registry.hpp"

namespace MatrixTransform {

    namespace { 
        MultiplierRegistrar CUDARegistrar( 
            "CUDA",
            []() -> std::unique_ptr<IMultiplier> { 
                return std::make_unique<CUDAMultiplier>();
            }
        );
    }

    #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess){
            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
        }
    }

    template <int TILE_WIDTH>
    __global__ void matmul_kernel(float* d_C, const float* d_A, const float* d_B, int rowsA, int colsA, int colsB) {
        
        int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
        int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

        __shared__ float tile_A[TILE_WIDTH][TILE_WIDTH];
        __shared__ float tile_B[TILE_WIDTH][TILE_WIDTH];

        float sum;

        for (int k = 0; k < colsA / TILE_WIDTH; ++k){
            int Ax = blockIdx.y * TILE_WIDTH + threadIdx.y;
            int Ay = k * TILE_WIDTH + threadIdx.x;

            int Bx = k * TILE_WIDTH + threadIdx.y;
            int By = blockIdx.x * TILE_WIDTH + threadIdx.x;

            tile_A[threadIdx.y][threadIdx.x] = d_A[Ax * colsA + Ay];
            tile_B[threadIdx.y][threadIdx.x] = d_B[Bx * colsB + By];

            __syncthreads();

            for (int k = 0; k < TILE_WIDTH; ++k) {
                sum += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];
            }

            __syncthreads();
        }

        d_C[row * colsB + col] = sum;
    }
        
    Matrix CUDAMultiplier::multiply(const Matrix& a, const Matrix& b) {
        Logger::getInstance().log(LogLevel::Debug, "Starting CUDA multiplication.");

        if (a.cols() != b.rows()) {
            Logger::getInstance().log(LogLevel::Error, "Matrix dimension mismatch.");
            throw std::invalid_argument("Matrix dimensions do not match for multiplication.");
        }

        Matrix c(a.rows(), b.cols());

        float* d_A, *d_B, *d_C; 
        size_t d_A_bytes = a.rows() * a.cols() * sizeof(float);
        size_t d_B_bytes = b.rows() * b.cols() * sizeof(float);
        size_t d_C_bytes = c.rows() * c.cols() * sizeof(float);
        
        Logger::getInstance().log(LogLevel::Debug, "Allocating memory on GPU...");
        gpuErrchk(hipMalloc(&d_A, d_A_bytes));
        gpuErrchk(hipMalloc(&d_B, d_B_bytes));
        gpuErrchk(hipMalloc(&d_C, d_C_bytes));

        Logger::getInstance().log(LogLevel::Debug, "Copying memory to GPU...");
        gpuErrchk(hipMemcpy(d_A, a.data(), d_A_bytes, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_B, b.data(), d_B_bytes, hipMemcpyHostToDevice));

        Logger::getInstance().log(LogLevel::Debug, "Zeroing out device result matrix C...");
        gpuErrchk(hipMemset(d_C, 0, d_C_bytes));

        constexpr int TILE_WIDTH = 16;
        dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
        dim3 numBlocks( (c.cols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
                        (c.rows() + threadsPerBlock.y - 1) / threadsPerBlock.y );

        Logger::getInstance().log(LogLevel::Debug, "Launching matmul kernel...");
        matmul_kernel<32><<<numBlocks, threadsPerBlock>>>(d_C, d_A, d_B, a.rows(), a.cols(), b.cols());

        gpuErrchk(hipGetLastError());
        gpuErrchk(hipDeviceSynchronize()); 

        Logger::getInstance().log(LogLevel::Debug, "Copying result matrix from GPU...");
        gpuErrchk(hipMemcpy(c.data(), d_C, d_C_bytes, hipMemcpyDeviceToHost));

        Logger::getInstance().log(LogLevel::Debug, "Freeing GPU memory...");
        gpuErrchk(hipFree(d_A));
        gpuErrchk(hipFree(d_B));
        gpuErrchk(hipFree(d_C));

        Logger::getInstance().log(LogLevel::Info, "CUDA multiplication complete.");
        return c;
    }

} // namespace MatrixTransform

// #endif