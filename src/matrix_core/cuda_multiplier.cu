#include "hip/hip_runtime.h"
// #ifdef WITH_CUDA

#include <Eigen/Dense>
#include <iostream>
#include <hip/hip_runtime.h>
#include "logger.hpp"
#include "cuda_multiplier.hpp"
#include "multiplier_registry.hpp"

namespace MatrixTransform {

    namespace { 
        MultiplierRegistrar CUDARegistrar( 
            "CUDA",
            []() -> std::unique_ptr<IMultiplier> { 
                return std::make_unique<CUDAMultiplier>();
            }
        );
    }

    #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess){
            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
        }
    }

    template <int TILE_WIDTH>
    __global__ void matmul_kernel(float* d_C, const float* d_A, const float* d_B, int rowsA, int colsA, int colsB) {
        int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
        int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

        __shared__ float tile_A[TILE_WIDTH][TILE_WIDTH];
        __shared__ float tile_B[TILE_WIDTH][TILE_WIDTH];

        float sum = 0.0f;

        for (int tile_num = 0; tile_num < (colsA + TILE_WIDTH - 1) / TILE_WIDTH; ++tile_num) {

            int source_A_row = row;
            int source_A_col = tile_num * TILE_WIDTH + threadIdx.x;

            int source_B_row = tile_num * TILE_WIDTH + threadIdx.y;
            int source_B_col = col;

            if (source_A_row < rowsA && source_A_col < colsA) {
                tile_A[threadIdx.y][threadIdx.x] = d_A[source_A_col * rowsA + source_A_row];
            } else {
                tile_A[threadIdx.y][threadIdx.x] = 0.0f;
            }

            if (source_B_row < colsA && source_B_col < colsB) {
                tile_B[threadIdx.y][threadIdx.x] = d_B[source_B_col * colsA + source_B_row];
            } else {
                tile_B[threadIdx.y][threadIdx.x] = 0.0f;
            }

            __syncthreads();

            for (int inner_k = 0; inner_k < TILE_WIDTH; ++inner_k) {
                sum += tile_A[threadIdx.y][inner_k] * tile_B[inner_k][threadIdx.x];
            }

            __syncthreads();
        }

        if (row < rowsA && col < colsB) {
            d_C[col * rowsA + row] = sum;
        }
    }
        
    Matrix CUDAMultiplier::multiply(const Matrix& a, const Matrix& b) {
        Logger::getInstance().log(LogLevel::Debug, "Starting CUDA multiplication.");

        if (a.cols() != b.rows()) {
            Logger::getInstance().log(LogLevel::Error, "Matrix dimension mismatch.");
            throw std::invalid_argument("Matrix dimensions do not match for multiplication.");
        }

        Matrix c(a.rows(), b.cols());

        float* d_A, *d_B, *d_C; 
        size_t d_A_bytes = a.rows() * a.cols() * sizeof(float);
        size_t d_B_bytes = b.rows() * b.cols() * sizeof(float);
        size_t d_C_bytes = c.rows() * c.cols() * sizeof(float);
        
        Logger::getInstance().log(LogLevel::Debug, "Allocating memory on GPU...");
        gpuErrchk(hipMalloc(&d_A, d_A_bytes));
        gpuErrchk(hipMalloc(&d_B, d_B_bytes));
        gpuErrchk(hipMalloc(&d_C, d_C_bytes));

        Logger::getInstance().log(LogLevel::Debug, "Copying memory to GPU...");
        gpuErrchk(hipMemcpy(d_A, a.data(), d_A_bytes, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_B, b.data(), d_B_bytes, hipMemcpyHostToDevice));

        Logger::getInstance().log(LogLevel::Debug, "Zeroing out device result matrix C...");
        gpuErrchk(hipMemset(d_C, 0, d_C_bytes));

        constexpr int TILE_WIDTH = 16;
        dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
        dim3 numBlocks( (c.cols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
                        (c.rows() + threadsPerBlock.y - 1) / threadsPerBlock.y );

        Logger::getInstance().log(LogLevel::Debug, "Launching matmul kernel...");
        matmul_kernel<TILE_WIDTH><<<numBlocks, threadsPerBlock>>>(d_C, d_A, d_B, a.rows(), a.cols(), b.cols());

        gpuErrchk(hipGetLastError());
        gpuErrchk(hipDeviceSynchronize()); 

        Logger::getInstance().log(LogLevel::Debug, "Copying result matrix from GPU...");
        gpuErrchk(hipMemcpy(c.data(), d_C, d_C_bytes, hipMemcpyDeviceToHost));

        gpuErrchk(hipDeviceSynchronize());

        Logger::getInstance().log(LogLevel::Debug, "Freeing GPU memory...");
        gpuErrchk(hipFree(d_A));
        gpuErrchk(hipFree(d_B));
        gpuErrchk(hipFree(d_C));

        Logger::getInstance().log(LogLevel::Info, "CUDA multiplication complete.");
        return c;
    }

} // namespace MatrixTransform

// #endif